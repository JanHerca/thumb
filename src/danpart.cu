
#include <hip/hip_runtime.h>
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

 /* This example demonstrates how to use the Cuda OpenGL bindings with the
  * runtime API.
  * Device code.
  */

//#ifndef _SIMPLEGL_KERNEL_H_
//#define _SIMPLEGL_KERNEL_H_

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////

extern "C"
__global__ void danpart(float4* pos, float * pdata, unsigned int width,
unsigned int height, int max_age, float time, float r1, float r2, float r3)
{
	// r1,r2,r3 curently not used
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned int arrayLoc = y*width*4 + x*4;
    unsigned int posLoc = y*width+x;

    if (pdata[arrayLoc] >= max_age)
       {
       pdata[arrayLoc] = 0;
       pdata[arrayLoc+1] = 0.002 * (sin(time) + (float)x / (float)width/10.0f ) ;

       pdata[arrayLoc+2] = 0;
       pdata[arrayLoc+3] = 0.002 * (cos(time) + (float)y / (float)height/10.0f );

       // maybe move the generation point around?

       pos[posLoc].x = 0;
       pos[posLoc].y = .5;
       pos[posLoc].z = 0;
       }
      
       float newX = pos[posLoc].x + pdata[arrayLoc+1];
       float newY = pos[posLoc].y + pdata[arrayLoc+2];
       float newZ = pos[posLoc].z + pdata[arrayLoc+3];

       pdata[arrayLoc] += 1;        // increase age
       pdata[arrayLoc+2] -= 0.0001; // gravity

       // tabletop surface
       if ((newY <= 0) && fabs(pos[posLoc].x)<5 && fabs(pos[posLoc].z)<5)
           {
           pdata[arrayLoc+2] = -0.7 * pdata[arrayLoc+2];
           }

      // now need to modify the color info in the array
//      pos[width*height + posLoc].x = 0.0f;//red
//      pos[width*height + posLoc].y = 1.0f;//green
//      pos[width*height + posLoc].z = 0.0f;//blue
	float colorFreq = 16.0f;
    pos[width*height + posLoc].y = (cos(colorFreq * 2.0 * pdata[arrayLoc]/max_age))/2.0f + 0.5f ;
    pos[width*height + posLoc].x = (cos(colorFreq * 1.0 * pdata[arrayLoc]/max_age))/2.0f + 0.5f ;
    pos[width*height + posLoc].z = (cos(colorFreq * 4.0 * pdata[arrayLoc]/max_age))/2.0f + 0.5f ;


    // write output vertex
     pos[posLoc] = make_float4(newX, newY, newZ, 1.0f);
}


//#endif // #ifndef _SIMPLEGL_KERNEL_H_
