#include "hip/hip_runtime.h"
/* 
dan Sandin 8-22-10
*/

#include "danglobs.cpp"
__constant__ float  refldata[REFL_DATA_MUNB][REFL_DATA_ROWS][REFL_DATA_ROW_ELEM];
__constant__ float  injdata[INJT_DATA_MUNB][INJT_DATA_ROWS][INJT_DATA_ROW_ELEM];


///////////////////////////////////////////////////////////////////////////////
//! Simple partical system
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////

__device__ void  injector0(unsigned int arrayLoc,unsigned int posLoc,float time,float4* pos, float* pdata){
	//ovels
	//  sin(time) + x index/width, x  y are randomly selected because of randon nature of age
	// x afects angular velocity distribution x,y afects liniar velocity distribution
//     pdata[arrayLoc+1] = 0.02 * (sin(time/5 + (float)x/(float)width/10.0) * (float)(x * y )/ (float)(width * height)/1.0f ) ;//x velocity  sin(time) + x index/width, x is randomly selected because of randon nature of age
     pdata[arrayLoc+1] = 0.02 * (sin(time/5  + (pdata[arrayLoc+5] + 1)/50) * ( ((pdata[arrayLoc+5]  +1)/1 ) * (pdata[arrayLoc+4] + 1.0)/1)  ) ;//x velocity  sin(time) + x index/width, x is randomly selected because of randon nature of age

     pdata[arrayLoc+2] = 0;
	//ovels       
//       	pdata[arrayLoc+3] = 0.02 * (cos(time/5 + (float)x/(float)width/10.0) * (float)(x *y) / (float)(width * height)/1.0f );// y velocity
       	pdata[arrayLoc+3] = 0.02 * (cos(time/5  + (pdata[arrayLoc+5] + 1)/50)  *( ((pdata[arrayLoc+5]  +1)/1 ) * (pdata[arrayLoc+4] + 1.0)/1));// y velocity


       // maybe move the generation point around?
		
       
			{
			pos[posLoc].x = 0;
       		pos[posLoc].y = 0.5;
       		pos[posLoc].z = 0;
			}
}
__device__ float  distRnd1( float seed, int iter){

	unsigned int rndint1;
	rndint1 = (unsigned int)(((seed +1.0)/2.0) *32768) % 32768;
	float sum ;
	sum =0;
	for ( int i = 0;i<iter;i++)
		{
		rndint1 = ((rndint1 * 1103515245 + 12345)/65536) % 32768;
		sum = sum +  0.0002 * (rndint1 % 10000) -1.0;
		}

	return	sum/iter;		
}

__device__ void  injector1(unsigned int arrayLoc,unsigned int posLoc,float time,float4* pos, float* pdata){
	float rnd1,rnd2,rnd3;
	rnd1 = distRnd1(pdata[arrayLoc+4] , 5);
	rnd2 = distRnd1(pdata[arrayLoc+5] , 5);
	rnd3 = distRnd1(pdata[arrayLoc+6] , 5);

	
	pdata[arrayLoc+1] = 0.02 * (sin(time/5  + (rnd1)/50) * (rnd2 +1)) ;//x vloocity 
    pdata[arrayLoc+2] = 0.002 * rnd3;
    pdata[arrayLoc+3] = 0.02 * (cos(time/5  + (rnd1)/50)  *(rnd2 +1));	//y volocity

	pos[posLoc].x = 0;
    //pos[posLoc].y = pdata[7];
    pos[posLoc].y = 0;
     pos[posLoc].z = 0;
			
}
__device__ void  injector2(unsigned int arrayLoc,unsigned int posLoc,int injNum,float time,float4* pos, float* pdata,float* debugData){
	float rnd1,rnd2,rnd3,rnd4,rnd5;
	float dt,du,dx,dy,dz,dx2,dy2,dz2,len,vx,vy,vz,dxt,dyt,dzt,dxu,dyu,dzu;
// float dv
/*
	injdata[injNum][1][0]// type, injection ratio ie streem volume, ~
	injdata[injNum][2][0];//x,y,z position
	injdata[injNum][3][0];//x,y,z velocity
	injdata[injNum][4][0];//x,y,z size
	injdata[injNum][5][0];//t,u,v jiter v not implimented = speed 
	injdata[injNum][6][0];//speed jiter ~ 
	injdata[injNum][7][0];//centrality of rnd distribution speed dt tu 

*/
	if ((pdata[arrayLoc+4] +1) /2 >  injdata[injNum][1][1]){ return;}// reterns without injection ?????
	
	rnd1 = (distRnd1(pdata[arrayLoc+4] , (int)injdata[injNum][7][0])+1)/2;
	rnd2 = (distRnd1(pdata[arrayLoc+5] , (int)injdata[injNum][7][1])+1)/2;
	rnd3 = (distRnd1(pdata[arrayLoc+6] , (int)injdata[injNum][7][2])+1)/2;
	rnd4 = (distRnd1(pdata[arrayLoc+4],1) );
	rnd5 = (distRnd1(pdata[arrayLoc+5],1) );
	//float rnd6 = (distRnd1(pdata[arrayLoc+6],1) );
	vx = injdata[injNum][3][0];vy = injdata[injNum][3][1];vz = injdata[injNum][3][2];//direction of spray	

	dt = injdata[injNum][5][0];du = injdata[injNum][5][1];// dv = injdata[injecti +17] * 0;// z component not implimented jitterelitive to direction of spreay

	// vector vx,vy,vz X 0,1,0
	dx = -vz;dy = 0;dz = vx;//  dt directon

	len = sqrt(dx*dx +dy*dy + dz*dz);
	if (len ==0)
		{
		dx = 0;dy =0;dz =0;
		}
	else{
		dx =dx/len;dy =dy/len;dz =dz/len;
		}
		//scale by dt amout of jitter in dt direction
		dxt = dx *dt;dyt = dy * dt;dzt = dz *dt;

	// vector vx,vy,vz X 0,1,0 X vx,vy,vz 
	dx2 = vy*vx;dy2 = vz*vz-vx*vx;dz2 = vy*vz;// du direction
	len = sqrt(dx2*dx2 +dy2*dy2 + dz2*dz2);
	if (len ==0)
		{
		dx2 = 0;dy2 =0;dz2 =0;
		}
	else{
		dx2 =dx2/len;dy2 =dy2/len;dz2 =dz2/len;
		}
		//scale by du amout of jutter in du direction
		dxu = dx2 *du;dyu = dy2 * du;dzu = dz2 *du;

	
	//indesices num injectors =0,position =6,velosity =9, size =12 tuv jiter = 15,speed = 18,centrality of randum  
	//         3 +             speed component                          velocity          t jitter u jitter

	pdata[arrayLoc+1] = ( rnd1 * injdata[injNum][6][0]) * (injdata[injNum][3][0] + dxt * rnd2 + dxu * rnd3) ;//x vloocity 
    pdata[arrayLoc+2] = ( rnd1  * injdata[injNum][6][0]) * (injdata[injNum][3][1] + dyt * rnd2+ dyu * rnd3) ; // y velocity
    pdata[arrayLoc+3] = ( rnd1  * injdata[injNum][6][0]) * (injdata[injNum][3][2] + dzt * rnd2+ dzu * rnd3);	//z volocity

	// size computation  xform  to dt du dv

	dt = injdata[injNum][4][0];du = injdata[injNum][4][1];//dv = injdata[injecti +14] * 0;//re use varables z component not implimented jitterelitive to direction of spreay
	dxt = dx *dt;dyt = dy * dt;dzt = dz *dt;
	dxu = dx2 *du;dyu = dy2 * du;dzu = dz2 *du;

	pos[posLoc].x = injdata[injNum][2][0] +  dxt * rnd4 + dxu * rnd5;
   
	pos[posLoc].y = injdata[injNum][2][1] + dyt * rnd4 + dyu * rnd5 ;
 
    pos[posLoc].z = injdata[injNum][2][2]  + dzt * rnd4+ dzu * rnd5;
if (DEBUG == 1)
	{
	
	int dbi =0;
	debugData[dbi + 0] = (float)injNum ;debugData[dbi + 1] =  injdata[injNum][1][1];debugData[dbi + 2] =0;
	dbi=dbi+3;
	debugData[dbi + 0] = dx;debugData[dbi + 1] = dy;debugData[dbi + 2] = dz;
	dbi=dbi+3;
	debugData[dbi + 0] = dx2;debugData[dbi + 1] = dy2;debugData[dbi + 2] = dz2;
	dbi=dbi+3;
	debugData[dbi + 0] = dxt;debugData[dbi + 1] = dyt;debugData[dbi + 2] = dzt;
	dbi=dbi+3;
	debugData[dbi + 0] = dxu;debugData[dbi + 1] = dyu;debugData[dbi + 2] = dzu;
	
	}
			
}
	__device__ void  planeReflector1(float posX,float posY,float posZ,float newX, float newY,float newZ,unsigned int arrayLoc,unsigned int posLoc,int reflNum,float time,float4* pos, float* pdata,float* debugData)
{
	float xn =1,yn =1,zn =0, rad =1,damping =.7;
	float xp,yp,zp;

	//indexices num injectors =0,position =[reflNum][1][0],normal =[reflNum][2][0], size =[reflNum][3][0] tuv jiter = [reflNum ][4][0],damping = [reflNum ][4][0],centrality of randum = 21

//dataorginization  refldata[reflNum][rownum][quardinare numbr x=0,1=y,2=z]
//dataorginization  type rownum 0 ~~ ,position 1,normal 2,radis 3,reflection coef 5,

	xn = refldata[reflNum][2][0];yn = refldata[reflNum][2][1];zn = refldata[reflNum][2][2];//normal
	rad = refldata[reflNum][3][0];
	damping = refldata[reflNum][5][0];
	xp = refldata[reflNum][1][0];yp = refldata[reflNum][1][1];zp = refldata[reflNum][1][2];//reflector position



	float length = sqrt(xn * xn + yn * yn + zn * zn);
    xn = xn/length;
    yn = yn/length;
    zn = zn/length;

	float distx = newX - xp;//point position - reflector position
	float disty = newY - yp;
	float distz = newZ - zp;


	 	float xv = pdata[arrayLoc+1];float yv = pdata[arrayLoc+2];float zv = pdata[arrayLoc+3];

	   	if ((fabs(distx) <= rad) && (fabs(disty)<= rad) && (fabs(distz) <= rad))
		{
	
	   		if ((distx * xn + disty * yn + distz * zn) <=0)

			{

		  		float ndotv = xv * xn + yv * yn + zv * zn;
				
	      		float newVX =(xv - 2.0*ndotv*xn);
		       	float newVY =(yv - 2.0*ndotv*yn);
		     	float newVZ =(zv - 2.0*ndotv*zn);
				// experments to lower traping  did not work
				//damping =1;
	      		//pos[posLoc].x  = posX + newVX;
       			//pos[posLoc].y  = posY + newVY;
       			//pos[posLoc].z  = posZ + newVZ;
	      		//pos[posLoc].x  = posX - (newX - posX) + newVX;
       			//pos[posLoc].y  = posY - (newY - posY) + newVY;
       			//pos[posLoc].z  = posZ - (newZ - posZ  + newVZ);
		      	pdata[arrayLoc+1] =newVX*damping;
		       	pdata[arrayLoc+2] =newVY*damping;
		     	pdata[arrayLoc+3] =newVZ*damping;
				
				//pdata[arrayLoc] = 0;// temp set age to 0
				
			}
	   }

}

extern "C"
__global__ void Point1(float4* pos, float * pdata,float * debugData ,unsigned int width,
unsigned int height, int max_age, float time, float r1, float r2, float r3)
{

	// r1,r2,r3 curently not used
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	
    unsigned int arrayLoc = y*width*7 + x*7;
    unsigned int posLoc = y*width+x;
	float newX,newY,newZ,posX,posY,posZ;

 /*
  	arrayLoc is data index of partical in pdata
    pdata [arrayLoc] = age ,pdata[arrayLoc +1 +2  +3 = x ,y,z velocity +rd1 rnd2 rnd3

    posLoc is index of partical location and[width*height + posLoc] index of color
    pos[posLoc].x .y .z  is x,y,z pos
    pos[ [width*height + posLoc].x .y .z is red green blue color
	In lines mode vbo is twice as big with tail and head psitions
*/
	

  	if (pdata[arrayLoc] >= max_age)
		{
		
		int injecNum = ((arrayLoc/7) % (int) injdata[0][0][0]) +1;// pdata row mod number of injectors 
		injector2(arrayLoc,posLoc,injecNum,time,pos,pdata,debugData);
		pdata[arrayLoc] = 0;//set age to 0

        }

		posX=pos[posLoc].x;posY=pos[posLoc].y;posZ=pos[posLoc].z;
 
  // add velocity to position  ie intigrate
		{
      	newX = posX + pdata[arrayLoc+1];
       	newY = posY + pdata[arrayLoc+2];
       	newZ = posZ + pdata[arrayLoc+3];
		}


       pdata[arrayLoc] += 1;        // increase age
       pdata[arrayLoc+2] -= 0.00001; // gravity

 
	// reflector

		for (int reflNum = 1;reflNum <= refldata[0][0][0]  ;reflNum ++)
			{
				
				//planeReflector1( pos[posLoc].x, pos[posLoc].y, pos[posLoc].z,arrayLoc,posLoc,reflNum,time,pos,pdata,debugData);			
				planeReflector1(posX,posY,posZ, newX, newY, newZ,arrayLoc,posLoc,reflNum,time,pos,pdata,debugData);			
			}
	
 
	
	float colorFreq = 16.0f;
 
			{
   	 		pos[width*height + posLoc].y = (cos(colorFreq * 2.0 * pdata[arrayLoc]/max_age))/2.0f + 0.5f ;//green
    		pos[width*height + posLoc].x = (cos(colorFreq * 1.0 * pdata[arrayLoc]/max_age))/2.0f + 0.5f ;//red
    		pos[width*height + posLoc].z = (cos(colorFreq * 4.0 * pdata[arrayLoc]/max_age))/2.0f + 0.5f ;//blue
    		// write output vertex
    		 pos[posLoc] = make_float4(newX, newY, newZ, 1.0f);
			}

}
extern "C"
__global__ void PointSquars(float4* pos, float * pdata, unsigned int width,
unsigned int height, int max_age, float time, float r1, float r2, float r3)
{

	// r1,r2,r3 curently not used
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    unsigned int arrayLoc = y*width*4 + x*4;
    unsigned int posLoc = y*width+x;
	float newX,newY,newZ;

 /*
  	arrayLoc is data index of partical in pdata
    pdata [arrayLoc] = age ,pdata[arrayLoc +1 +2  +3 = x ,y,z velocity

    posLoc is index of partical location and[width*height + posLoc] index of color
    pos[posLoc].x .y .z  is x,y,z pos
    pos[ [width*height + posLoc].x .y .z is red green blue color
	In lines mode vbo is twice as big with tail and head psitions
*/

	if (pdata[arrayLoc] >= max_age)
		{
		pdata[arrayLoc] = 0;
		
      	 pdata[arrayLoc+1] = 0.002 * (sin(time) + (float)x / (float)width/10.0f ) ;//x velocity  sin(time) + x index/width, x is randomly selected because of randon nature of age


       	pdata[arrayLoc+2] = 0;
		
     	 pdata[arrayLoc+3] = 0.002 * (cos(time) + (float)(y) / (float)( height)/10.0f );// y velocity
	
		

       // maybe move the generation point around?
		

			{
			pos[posLoc].x = 0;
       		pos[posLoc].y = 0.5;
       		pos[posLoc].z = 0;
			}

       }

      // add velocity to position  ie intigrate
 			{
      		newX = pos[posLoc].x + pdata[arrayLoc+1];
       		newY = pos[posLoc].y + pdata[arrayLoc+2];
       		newZ = pos[posLoc].z + pdata[arrayLoc+3];
			}


       pdata[arrayLoc] += 1;        // increase age
       pdata[arrayLoc+2] -= 10.1; // gravity

       // check aganst tabletop surface reverse velocity
 			{

       		if ((newY <= 0) && fabs(pos[posLoc].x)<5 && fabs(pos[posLoc].z)<5)
           		{
           		//pdata[arrayLoc+2] = -0.7 * pdata[arrayLoc+2];
           		}
  			}



      // now need to modify the color info in the array
//      pos[width*height + posLoc].x = 0.0f;//red
//      pos[width*height + posLoc].y = 1.0f;//green
//      pos[width*height + posLoc].z = 0.0f;//blue
	float colorFreq = 16.0f;

			{
   	 		pos[width*height + posLoc].y = (cos(colorFreq * 2.0 * pdata[arrayLoc]/max_age))/2.0f + 0.5f ;
    		pos[width*height + posLoc].x = (cos(colorFreq * 1.0 * pdata[arrayLoc]/max_age))/2.0f + 0.5f ;
    		pos[width*height + posLoc].z = (cos(colorFreq * 4.0 * pdata[arrayLoc]/max_age))/2.0f + 0.5f ;
    		// write output vertex
    		 pos[posLoc] = make_float4(newX, newY, newZ, 1.0f);
			}


}





